
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>

// __global__ void merge(float *data, float *work, int k)
// {
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
    
//   int l = index * k;
//   int m = l + k / 2;
//   int r = l + k;

//   int first = l;
//   int second = m;

//   for (int i = l; i < r; i++)
//   {
//       if (first < m && (second >= r || data[first] <= data[second]))
//       {
//           work[i] = data[first];
//           first += 1;
//       }
//       else
//       {
//           work[i] = data[second];
//           second += 1;
//       }
//   }
// }

// __global__ void parallel_merge(float *data, float *work)
// {
//   uint half = blockDim.x >> 1;
//   uint pos = blockIdx.x * blockDim.x;
//   uint left_array = threadIdx.x < half ? 1 : 0;

//   float cur = data[pos + threadIdx.x];
//   uint i = 0;
//   uint j = half;

//   if (left_array)
//   {
//     while (i < j)
//     {
//       uint mid = i + (j - i) / 2;
//       if (cur <= data[pos + half + mid])
//         j = mid;
//       else
//         i = mid + 1;
//     }
//     work[pos + threadIdx.x + i] = cur;
//   }
//   else
//   {
//     while (i < j)
//     {
//       uint mid = i + (j - i) / 2;
//       if (cur < data[pos + mid])
//         j = mid;
//       else
//         i = mid + 1;
//     }
//     work[pos + threadIdx.x - half + i] = cur;
//   }
// }

__global__ void parallel_merge(float *data, float *work, int stride)
{
  uint index = threadIdx.x * stride;
  uint pos = blockIdx.x * blockDim.x * stride;
  uint half = blockDim.x * stride >> 1;
  uint left_array = index < half ? 1 : 0;

  for (uint s = 0; s < stride; s++)
  {
    float cur = data[pos + index + s];
    uint i = 0;
    uint j = half;
    
    if (left_array)
    {
      while (i < j)
      {
        uint mid = i + (j - i) / 2;
        if (cur <= data[pos + half + mid])
          j = mid;
        else
          i = mid + 1;
      }
      work[pos + index + s + i] = cur;
    }
    else
    {
      while (i < j)
      {
        uint mid = i + (j - i) / 2;
        if (cur < data[pos + mid])
          j = mid;
        else
          i = mid + 1;
      }
      work[pos + index + s - half + i] = cur;
    }
  }
}


int cuda_sort(int number_of_elements, float *a)
{
  hipError_t cudaStatus;
	
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}
  
  float *d_a, *d_b;

  hipMalloc((void **)&d_a, number_of_elements * sizeof(float));
  hipMalloc((void **)&d_b, number_of_elements * sizeof(float));
  
  hipMemcpy(d_a, a, number_of_elements * sizeof(float), hipMemcpyHostToDevice);
  
  int level = 0;
  for (int k = 2; k <= number_of_elements; k = 2 * k)
  {
    int num_merges = number_of_elements / k;
    if (k <= 1024)
    {
      if (level % 2 == 0)
        parallel_merge<<<num_merges, k>>>(d_a, d_b, 1);
      else
        parallel_merge<<<num_merges, k>>>(d_b, d_a, 1);
    }
    else
    {
      if (level % 2 == 0)
        parallel_merge<<<num_merges, 1024>>>(d_a, d_b, k / 1024);
      else
        parallel_merge<<<num_merges, 1024>>>(d_b, d_a, k / 1024);
    }
    hipDeviceSynchronize();
    level += 1;
  }

  if (level % 2 == 0)
    hipMemcpy(a, d_a, number_of_elements * sizeof(float), hipMemcpyDeviceToHost);
  else
    hipMemcpy(a, d_b, number_of_elements * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  return 0;
}